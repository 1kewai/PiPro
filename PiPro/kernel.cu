#include "hip/hip_runtime.h"
﻿//CUDA Cのためのinclude(gpu側)
#include "hip/hip_runtime.h"
#include ""

//C include(cpu側のみ)
#include <stdio.h>
#include <stdlib.h>

//CUDA How-to memo
//スレッドの固有の番号を取得:blockIdx.x * blockDim.x + threadIdx.x
//メモリの内容をコピー:hipMemcpy(HostMemory, Device_array, num_of_rnd * 4, hipMemcpyDeviceToHost);
//CUDAの処理完了を待機:hipDeviceSynchronize();
//デバイスメモリ確保:int* Device_addr; hipMalloc((void**)&Device_addr, サイズ);
//デバイスメモリ解放;hipFree(ポインタ)
//メモリコピー:hipMemcpy(HostMemory(dst), Device_array(src), num_of_rnd * 4, hipMemcpyDeviceToHost);


__device__ double Leibniz_One(unsigned long count) {
    count++;
    return 8.0 / (16.0 * count * (count - 1.0) + 3.0);
}

__global__ void Leibniz_to_Array(double *dst, unsigned int head) {
    dst[blockIdx.x * blockDim.x + threadIdx.x + head] = Leibniz_One(blockIdx.x * blockDim.x + threadIdx.x + head);
}

double Host_Leibniz(int start, int size,int Acc, double* Host_Leibniz_Array) {
    printf("starting %d\n", start);
    //ライプニッツの式を計算する際の各項を保存するためのメモリ確保
    //GPUデバイス側
    double* Device_Leibniz;
    hipMalloc((void**)&Device_Leibniz, Acc * 8);
    
    //ライップニッツの式を計算
    Leibniz_to_Array << <16, Acc / 16 >> > (Device_Leibniz, start);
    hipDeviceSynchronize();

    //GPUメモリから計算結果を転送
    hipMemcpy(Host_Leibniz_Array, Device_Leibniz, Acc * 8, hipMemcpyDeviceToHost);

    //cpuで足し算して出力
    double result = 0.0;
    for (int i = 0; i < Acc; i++) {
        result += Host_Leibniz_Array[i];
    }
    return  result;
 }

int main() {
    //ライプニッツの式を計算する際のGPUで一度に計算させる項数
    unsigned const int Acc=4096*4;
    //ライプニッツの式をGPUに計算させる回数
    unsigned const int count = 8;
    //ホスト側メモリ確保
    //中身はheapに確保する(stack overflow対策)
    double* Host_Leibniz_Array[count];
    //countの回数だけ処理を実行
    double re = 0;
    for (int i = 0; i < count; i++) {
        Host_Leibniz_Array[i] = new double[Acc];//heapに確保
        re += Host_Leibniz(i*Acc, Acc, Acc, Host_Leibniz_Array[i]);
    }
    printf("%1.10lf\n", re);
}